#include "hip/hip_runtime.h"
#include "CudaChecks.cuh"
#include "LayerClasses.cuh"
#include <cstdlib>
#include <iostream>
#include <random>
#include <stdexcept>
#define SHARED_BLOCK_SIZE BLOCK_SIZE + 2
#define POOL_BLOCK_SIZE 16
#define KERNEL_SIZE 3

convLayer::convLayer(ImageSize inputImageSize, ImageSize outputImageSize,
                     ImageSize kernelSize, uint8_t input_channels,
                     uint8_t output_channels)
    : input_channels(input_channels), output_channels(output_channels),
      HA(inputImageSize.height), WA(inputImageSize.width),
      HB(outputImageSize.height), WB(outputImageSize.width),
      HC(kernelSize.height), WC(kernelSize.width) {

  // Random number generator setup
  std::default_random_engine gen;
  float stddev =
      sqrtf(2.0f / (inputImageSize.height *
                    inputImageSize.width)); // He initialization for ReLU
  std::normal_distribution<float> dist(0.0f, stddev);
  kernels = (float *)malloc(output_channels * input_channels * KERNEL_SIZE *
                            KERNEL_SIZE * sizeof(float));
  for (int oc = 0; oc < output_channels; ++oc) {
    for (int ic = 0; ic < input_channels; ++ic) {
      for (int i = 0; i < KERNEL_SIZE * KERNEL_SIZE; ++i) {
        kernels[((oc * input_channels + ic) * KERNEL_SIZE * KERNEL_SIZE) + i] =
            dist(gen);
      }
    }
  }
  cudaCheck(hipMalloc((void **)&d_kernels, input_channels * output_channels *
                                                KERNEL_SIZE * KERNEL_SIZE *
                                                sizeof(float)));

  cudaCheck(hipMemcpy(d_kernels, kernels,
                       input_channels * output_channels * KERNEL_SIZE *
                           KERNEL_SIZE * sizeof(float),
                       hipMemcpyHostToDevice));
}

convLayer::~convLayer() {
  free(kernels);
  hipFree(d_kernels);
  // cudaCheck(hipFree(d_input_image));
  // cudaCheck(hipFree(d_output_image));
}

float *convLayer::forward(float *d_input_image, float *d_output_image) {

  int tile_output_width = BLOCK_SIZE - WC + 1;
  int tile_output_height = BLOCK_SIZE - HC + 1;

  int grid_x = (WB + tile_output_width - 1) / tile_output_width;
  int grid_y = (HB + tile_output_height - 1) / tile_output_height;

  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid(grid_x, grid_y, output_channels);

  int shared_height = BLOCK_SIZE + HC - 1;
  int shared_width = BLOCK_SIZE + WC - 1;

  int tile_width = WC + 1;  // adjust this based on needed coverage
  int tile_height = HC + 1; // adjust this based on needed coverage
  int shared_mem_bytes = tile_width * tile_height * sizeof(float);

  int total_outputs = output_channels * HB * WB;
  int threads_per_block = 256;
  int num_blocks = (total_outputs + threads_per_block - 1) / threads_per_block;

  Convolution3D_1d_launch<<<num_blocks, threads_per_block>>>(
      d_input_image, d_output_image, d_kernels, HA, WA, HB, WB, HC, WC,
      input_channels, output_channels);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cerr << hipGetErrorString(error) << std::endl;
    throw std::runtime_error("Cuda kernel failed\n");
  }

  cudaCheck(hipDeviceSynchronize());
  return d_output_image;
}

void convLayer::ReLU(float *B) {
  int total_elements = output_channels * WB * HB;
  int threads_per_block = 256;
  int blocks_per_grid =
      (total_elements + threads_per_block - 1) / threads_per_block;

  ReLU_kernel<<<blocks_per_grid, threads_per_block>>>(B, HB, WB,
                                                      output_channels);
  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cerr << hipGetErrorString(error) << std::endl;
    throw std::runtime_error("Cuda kernel failed\n");
  }

  cudaCheck(hipDeviceSynchronize());
}

maxPool::maxPool(int HA, int WA, int HB, int WB, int input_channels)
    : HA(HA), WA(WA), HB(HB), WB(WB), input_channels(input_channels) {}

float *maxPool::forward(float *d_input, float *d_output, int *d_max_ind) {
  int total_outputs = HB * WB * input_channels;
  int block_size = POOL_BLOCK_SIZE * POOL_BLOCK_SIZE;
  int grid_size = (total_outputs + block_size - 1) / block_size;

  maxPool2D<<<grid_size, block_size>>>(d_input, d_output, HA, WA, HB, WB,
                                       input_channels);
  cudaCheck(hipPeekAtLastError());
  cudaCheck(hipDeviceSynchronize());
  return d_output;
}

mlpLayer::mlpLayer(int input_size, int output_size)
    : input_size(input_size), output_size(output_size) {

  // Random number generator setup
  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> pos_dist(0.0f, 1.0f);
  std::uniform_real_distribution<float> neg_dist(-1.0f, 0.0f);

  // Allocate host memory
  bias = (float *)malloc(output_size * sizeof(float));
  weights = (float *)malloc(output_size * input_size * sizeof(float));

  // Initialize bias values
  for (int i = 0; i < output_size; ++i) {
    bias[i] = (i % 2 == 0) ? pos_dist(gen) : neg_dist(gen);
  }

  float stddev = sqrtf(2.0f / input_size);
  std::normal_distribution<float> dist(0.0f, stddev);
  for (int i = 0; i < input_size * output_size; ++i) {
    weights[i] = dist(gen);
  }

  // Allocate device memory
  cudaCheck(hipMalloc((void **)&d_bias, output_size * sizeof(float)));
  cudaCheck(hipMalloc((void **)&d_weights,
                       input_size * output_size * sizeof(float)));
  cudaCheck(
      hipMalloc((void **)&dL_dW, input_size * output_size * sizeof(float)));
  cudaCheck(hipMalloc((void **)&dL_db, output_size * sizeof(float)));
  cudaCheck(hipMalloc((void **)&dL_dx, input_size * sizeof(float)));
  cudaCheck(hipMalloc((void **)&dL_dz, output_size * sizeof(float)));
  cudaCheck(hipMalloc((void **)&dy_dz, output_size * sizeof(float)));
  cudaCheck(hipMalloc((void **)&d_weights_transpose,
                       input_size * output_size * sizeof(float)));

  // Copy host memory to device memory
  cudaCheck(hipMemcpy(d_bias, bias, output_size * sizeof(float),
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_weights, weights,
                       input_size * output_size * sizeof(float),
                       hipMemcpyHostToDevice));
}

mlpLayer::~mlpLayer() {
  free(bias);
  free(weights);
  cudaCheck(hipFree(d_bias));
  cudaCheck(hipFree(d_weights));
  cudaCheck(hipFree(d_weights_transpose));
  cudaCheck(hipFree(dL_dW));
  cudaCheck(hipFree(dL_dx));
  cudaCheck(hipFree(dL_dz));
  cudaCheck(hipFree(dy_dz));
}

float *mlpLayer::forward(float *d_input, float *d_output) {
  // dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE, 1);
  // dim3 DimGrid((output_size + BLOCK_SIZE - 1) / BLOCK_SIZE, 1);

  const int block_size = 16;
  dim3 DimBlock(BLOCK_SIZE * BLOCK_SIZE); // 1D thread block
  int tilesX = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int tilesY = (1 + BLOCK_SIZE - 1) / BLOCK_SIZE; // since HA == 1
  dim3 DimGrid(tilesX * tilesY);                  // 1D grid
  // Launch sgemm
  sgemm_1d<block_size><<<DimGrid, DimBlock>>>(
      d_input,     // A: input (1 x input_size)
      d_weights,   // B: weights (input_size x output_size)
      d_output,    // C: output (1 x output_size)
      1,           // HA
      input_size,  // WA
      input_size,  // HB
      output_size, // WB
      1,           // HC
      output_size  // WC
  );
  (hipDeviceSynchronize());

  // add the bias
  vecAdd<<<(output_size + 255) / 256, 256>>>(d_output, d_bias, false,
                                             output_size);
  cudaCheck(hipDeviceSynchronize());
  return d_output;
}

void mlpLayer::ReLU(float *d_input) {

  int threadsPerBlock = 256;
  int blocksPerGrid = (input_size + threadsPerBlock - 1) / threadsPerBlock;
  ReLU_kernel<<<blocksPerGrid, threadsPerBlock>>>(d_input, input_size);
  cudaCheck(hipDeviceSynchronize());
}
void mlpLayer::computeGradients(float *d_input, float *dL_dy) { // compute dy_dz
  int threadsPerBlock = 256;
  int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;
  reluBackward<<<blocksPerGrid, threadsPerBlock>>>(d_input, dy_dz, dL_dy,
                                                   output_size);
  cudaCheck(hipDeviceSynchronize());

  // elementwise operation of dL_dy and dy_dz to compute dL_dz
  tensorElementwiseMult<<<blocksPerGrid, threadsPerBlock>>>(dL_dy, dy_dz, dL_dz,
                                                            output_size);
  cudaCheck(hipDeviceSynchronize());

  // compute dL_dW
  constexpr int block_size = 16;
  threadsPerBlock = 256;
  blocksPerGrid =
      (input_size * output_size + threadsPerBlock - 1) / threadsPerBlock;
  sgemm_1d<block_size><<<blocksPerGrid, threadsPerBlock>>>(
      d_input, dL_dz, dL_dW, input_size, 1, 1, output_size, input_size,
      output_size);

  cudaCheck(hipDeviceSynchronize());

  // compute dL_db
  cudaCheck(hipMemcpy(dL_db, dL_dz, output_size * sizeof(float),
                       hipMemcpyDeviceToDevice));

  // compute dL_dx
  // First, transpose weights;
  transposeKernel<<<blocksPerGrid, threadsPerBlock>>>(
      d_weights, d_weights_transpose, input_size, output_size);
  cudaCheck(hipDeviceSynchronize());

  // Now compute sgemm of W^T @ dL_dz
  blocksPerGrid = (input_size + threadsPerBlock - 1) / threadsPerBlock;
  sgemm_1d<block_size><<<blocksPerGrid, threadsPerBlock>>>(
      dL_dz, d_weights_transpose, dL_dx, 1, output_size, output_size,
      input_size, 1, input_size);

  cudaCheck(hipDeviceSynchronize());
}

float *mlpLayer::backProp(float *d_input, float *dL_dy, float alpha) {
  computeGradients(d_input, dL_dy);

  // matAdd all the gradients
  bool neg = true; // subtract all gradients

  // first do dL_dW
  int threadsPerBlock = 256;
  int blocksPerGrid =
      (output_size * input_size + threadsPerBlock - 1) / threadsPerBlock;
  matAdd<<<blocksPerGrid, threadsPerBlock>>>(
      d_weights, dL_dW, d_weights, input_size, output_size, neg, alpha);

  cudaCheck(hipDeviceSynchronize());

  // now subtract bias gradient
  blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;
  matAdd<<<blocksPerGrid, threadsPerBlock>>>(d_bias, dL_db, d_bias, 1,
                                             output_size, neg, alpha);

  cudaCheck(hipDeviceSynchronize());

  return dL_dx; // return input_grad pointer for next layer to use
}

float *mlpLayer::getHostWeights(){
  return weights;
}

float *mlpLayer::getHostBias(){
  return bias;
}

float *mlpLayer::getDeviceWeights(){
  return d_weights;
}

float *mlpLayer::getDeviceBias(){
  return d_bias;
}

float *mlpLayer::getWeightGrad(){
  return dL_dW;
}


float *mlpLayer::getInputGrad(){
  return dL_dx;
}

float *mlpLayer::getBiasGrad(){
  return dL_db;
}

float *mlpLayer::getOutputGrad(){
  return dL_dz;
}

SoftmaxLayer::SoftmaxLayer(int input_size, int output_size)
    : input_size(input_size), output_size(output_size) {
  cudaCheck(hipHostAlloc(&h_loss, sizeof(float), hipHostMallocDefault));
  cudaCheck(
      hipHostAlloc(&y_hat, sizeof(float) * output_size, hipHostMallocDefault));
  cudaCheck(
      hipHostAlloc(&y, sizeof(float) * output_size, hipHostMallocDefault));
  cudaCheck(hipMalloc(&d_loss, sizeof(float)));
}

SoftmaxLayer::~SoftmaxLayer() {
  cudaCheck(hipHostFree(h_loss));
  cudaCheck(hipHostFree(y_hat));
  cudaCheck(hipHostFree(y));
  cudaCheck(hipFree(d_loss));
}

void SoftmaxLayer::softMax(float *d_input, float *d_output) {
  int blockSize = 128;
  int gridSize = (output_size + blockSize - 1) / blockSize;
  softmaxKernel<<<gridSize, blockSize>>>(d_input, d_output, output_size);
}

float SoftmaxLayer::computeLoss(float *d_y_hat, float *d_y) {
  cudaCheck(hipMemcpy(y_hat, d_y_hat, sizeof(float) * output_size,
                       hipMemcpyDeviceToHost));
  cudaCheck(
      hipMemcpy(y, d_y, sizeof(float) * output_size, hipMemcpyDeviceToHost));
  *h_loss = computeCrossEntropyLoss(y_hat, y, output_size);
  hipMemcpy(d_loss, h_loss, sizeof(float), hipMemcpyHostToDevice);
  return *h_loss;
}

float *SoftmaxLayer::backProp(float *d_y_hat, float *d_y, float alpha) {

  int threadsPerBlock = 256;
  int blocksPerGrid = (output_size + threadsPerBlock - 1) / threadsPerBlock;
  dim3 blockDim(threadsPerBlock);
  dim3 DimGrid(blocksPerGrid);
  vecAdd<<<DimGrid, DimGrid>>>(d_y_hat, d_y, true, output_size);
  cudaCheck(hipDeviceSynchronize());
  return d_y_hat;
}
