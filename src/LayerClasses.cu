#include "hip/hip_runtime.h"
#include "LayerClasses.cuh"
#include "cudaClasses.cuh"
#include "cudaKernels.cuh"
#include <cstdlib>
#include <iostream>
#include <random>
#include <stdexcept>

#define SHARED_BLOCK_SIZE BLOCK_SIZE + 2
#define POOL_BLOCK_SIZE 16
#define KERNEL_SIZE 3

Layer::~Layer() {}

// ConvLayer

ConvLayer::ConvLayer(ImageSize input_image_size, ImageSize output_image_size,
                     ImageSize kernel_size, uint8_t input_channels,
                     uint8_t output_channels)
    : input_channels(input_channels), output_channels(output_channels),
      ha(input_image_size.height), wa(input_image_size.width),
      hb(output_image_size.height), wb(output_image_size.width),
      hc(kernel_size.height), wc(kernel_size.width) {

  std::default_random_engine gen;
  float stddev =
      sqrtf(2.0f / (input_image_size.height * input_image_size.width));
  std::normal_distribution<float> dist(0.0f, stddev);

  kernels = (float *)malloc(output_channels * input_channels * KERNEL_SIZE *
                            KERNEL_SIZE * sizeof(float));

  for (int oc = 0; oc < output_channels; ++oc) {
    for (int ic = 0; ic < input_channels; ++ic) {
      for (int i = 0; i < KERNEL_SIZE * KERNEL_SIZE; ++i) {
        kernels[((oc * input_channels + ic) * KERNEL_SIZE * KERNEL_SIZE) + i] =
            dist(gen);
      }
    }
  }

  cuda_check(hipMalloc((void **)&d_kernels, input_channels * output_channels *
                                                 KERNEL_SIZE * KERNEL_SIZE *
                                                 sizeof(float)));

  cuda_check(hipMemcpy(d_kernels, kernels,
                        input_channels * output_channels * KERNEL_SIZE *
                            KERNEL_SIZE * sizeof(float),
                        hipMemcpyHostToDevice));
}

ConvLayer::~ConvLayer() {
  free(kernels);
  hipFree(d_kernels);
}

int ConvLayer::get_num_outputs() { return output_channels * hb * wb; }
int ConvLayer::get_num_inputs() { return input_channels * ha * wa; }

void ConvLayer::forward(float *d_input_image, float *d_output_image,
                        int batch_size) {
  int total_outputs = output_channels * hb * wb;
  int threads_per_block = 256;
  int num_blocks = (total_outputs + threads_per_block - 1) / threads_per_block;

  convolution3d_1d_launch<<<num_blocks, threads_per_block>>>(
      d_input_image, d_output_image, d_kernels, ha, wa, hb, wb, hc, wc,
      input_channels, output_channels);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cerr << hipGetErrorString(error) << std::endl;
    throw std::runtime_error("Cuda kernel failed\n");
  }

  cuda_check(hipDeviceSynchronize());
}

// TODO: Define this back_prop method. This is a place holder override for the
// Layer
//  virtual method
void ConvLayer::back_prop(float *d_input, float *d_grad_output, float alpha) {}

void ConvLayer::relu(float *b, int batch_size) {
  int total_elements = output_channels * wb * hb;
  int threads_per_block = 256;
  int blocks_per_grid =
      (total_elements + threads_per_block - 1) / threads_per_block;

  relu_kernel<<<blocks_per_grid, threads_per_block>>>(b, hb, wb,
                                                      output_channels);
  cuda_check(hipDeviceSynchronize());
}

float *ConvLayer::get_input_grad() { return dl_dx; }
// MaxPool

MaxPool::MaxPool(int ha, int wa, int hb, int wb, int input_channels)
    : ha(ha), wa(wa), hb(hb), wb(wb), input_channels(input_channels) {}

int MaxPool::get_num_outputs() { return input_channels * hb * wb; }
int MaxPool::get_num_inputs() { return input_channels * ha * wa; }

void MaxPool::forward(float *d_input, float *d_output, int batch_size) {}

void MaxPool::forward(float *d_input, float *d_output, int *d_max_ind,
                      int batch_size) {
  int total_outputs = hb * wb * input_channels;
  int block_size = POOL_BLOCK_SIZE * POOL_BLOCK_SIZE;
  int grid_size = (total_outputs + block_size - 1) / block_size;

  max_pool2d<<<grid_size, block_size>>>(d_input, d_output, ha, wa, hb, wb,
                                        input_channels);

  cuda_check(hipPeekAtLastError());
  cuda_check(hipDeviceSynchronize());
}

// TODO Define this method for conv layers
void MaxPool::back_prop(float *d_input, float *d_grad_output, float alpha) {}

float *MaxPool::get_input_grad() { return dl_dx; }

// MlpLayer

MlpLayer::MlpLayer(int input_size, int output_size)
    : input_size(input_size), output_size(output_size) {

  std::random_device rd;
  std::mt19937 gen(rd());
  std::uniform_real_distribution<float> pos_dist(0.0f, 1.0f);
  std::uniform_real_distribution<float> neg_dist(-1.0f, 0.0f);

  bias = (float *)malloc(output_size * sizeof(float));
  weights = (float *)malloc(output_size * input_size * sizeof(float));

  for (int i = 0; i < output_size; ++i) {
    bias[i] = (i % 2 == 0) ? pos_dist(gen) : neg_dist(gen);
  }

  float stddev = sqrtf(2.0f / input_size);
  std::normal_distribution<float> dist(0.0f, stddev);
  for (int i = 0; i < input_size * output_size; ++i) {
    weights[i] = dist(gen);
  }

  cuda_check(hipMalloc((void **)&d_bias, output_size * sizeof(float)));
  cuda_check(hipMalloc((void **)&d_weights,
                        input_size * output_size * sizeof(float)));
  cuda_check(
      hipMalloc((void **)&dl_dw, input_size * output_size * sizeof(float)));
  cuda_check(hipMalloc((void **)&dl_db, output_size * sizeof(float)));
  cuda_check(hipMalloc((void **)&dl_dx, input_size * sizeof(float)));
  cuda_check(hipMalloc((void **)&dl_dz, output_size * sizeof(float)));
  cuda_check(hipMalloc((void **)&dy_dz, output_size * sizeof(float)));
  cuda_check(hipMalloc((void **)&d_weights_transpose,
                        input_size * output_size * sizeof(float)));

  cuda_check(hipMemcpy(d_bias, bias, output_size * sizeof(float),
                        hipMemcpyHostToDevice));
  cuda_check(hipMemcpy(d_weights, weights,
                        input_size * output_size * sizeof(float),
                        hipMemcpyHostToDevice));
}

MlpLayer::~MlpLayer() {
  free(bias);
  free(weights);
  cuda_check(hipFree(d_bias));
  cuda_check(hipFree(d_weights));
  cuda_check(hipFree(d_weights_transpose));
  cuda_check(hipFree(dl_dw));
  cuda_check(hipFree(dl_dx));
  cuda_check(hipFree(dl_dz));
  cuda_check(hipFree(dy_dz));
  cuda_check(hipFree(dl_db));
}

int MlpLayer::get_num_outputs() { return output_size; }
int MlpLayer::get_num_inputs() { return input_size; }

void MlpLayer::forward(float *d_input, float *d_output, int batch_size) {
  const int block_size = 16;
  dim3 dim_block(BLOCK_SIZE * BLOCK_SIZE);
  int tiles_x = (output_size + BLOCK_SIZE - 1) / BLOCK_SIZE;
  int tiles_y = 1;
  dim3 dim_grid(tiles_x * tiles_y, 1, batch_size);

  sgemm_1d<block_size><<<dim_grid, dim_block>>>(d_input, d_weights, d_output, 1,
                                                input_size, input_size,
                                                output_size, 1, output_size);
  hipDeviceSynchronize();

  vec_add<<<(output_size + 255) / 256, 256>>>(d_output, d_bias, d_output, false,
                                              output_size);
  cuda_check(hipDeviceSynchronize());
  relu(d_output, batch_size);
}

void MlpLayer::relu(float *d_input, int batch_size) {
  int threads_per_block = 256;
  int total_elements = 1 * output_size * 1 * batch_size;

  int blocks_per_grid =
      (total_elements + threads_per_block - 1) / threads_per_block;

  relu_kernel<<<blocks_per_grid, threads_per_block>>>(d_input,
                                                      1,           // hb
                                                      output_size, // wb
                                                      1            // channels
  );
  cuda_check(hipDeviceSynchronize());
}

void MlpLayer::compute_gradients(float *d_input, float *dl_dy) {
  int threads_per_block = 256;
  int blocks_per_grid =
      (output_size + threads_per_block - 1) / threads_per_block;

  // 1) Compute dl_dz = dl_dy * relu'(z)  (relu_backward applies mask)
  relu_backward<<<blocks_per_grid, threads_per_block>>>(
      d_input, // input (used to compute mask input>0)
      dl_dy,   // grad_output = upstream gradient
      dl_dz,   // grad_input = result dl_dz
      output_size);
  cuda_check(hipDeviceSynchronize());

  // 2) Compute weight gradients: dl_dw = input^T * dl_dz
  constexpr int block_size = 16;
  threads_per_block = 256;
  blocks_per_grid =
      (input_size * output_size + threads_per_block - 1) / threads_per_block;
  sgemm_1d<block_size><<<blocks_per_grid, threads_per_block>>>(
      d_input, dl_dz, dl_dw, // A = input, B = dl_dz, C = dl_dw
      input_size, 1, 1, output_size, input_size, output_size);
  cuda_check(hipDeviceSynchronize());

  // 3) Copy bias grads (dl_db = dl_dz)
  cuda_check(hipMemcpy(dl_db, dl_dz, output_size * sizeof(float),
                        hipMemcpyDeviceToDevice));

  // 4) Transpose weights and compute input gradient dl_dx = dl_dz * W^T
  transpose_kernel<<<blocks_per_grid, threads_per_block>>>(
      d_weights, d_weights_transpose, input_size, output_size);
  cuda_check(hipDeviceSynchronize());

  blocks_per_grid = (input_size + threads_per_block - 1) / threads_per_block;
  sgemm_1d<block_size><<<blocks_per_grid, threads_per_block>>>(
      dl_dz, d_weights_transpose, dl_dx, 1, output_size, output_size,
      input_size, 1, input_size);
  cuda_check(hipDeviceSynchronize());
}

void MlpLayer::back_prop(float *d_input, float *dl_dy, float alpha) {
  compute_gradients(d_input, dl_dy);
  bool neg = true;

  int threads_per_block = 256;
  int blocks_per_grid =
      (output_size * input_size + threads_per_block - 1) / threads_per_block;
  mat_add<<<blocks_per_grid, threads_per_block>>>(
      d_weights, dl_dw, d_weights, input_size, output_size, neg, alpha);
  cuda_check(hipDeviceSynchronize());

  blocks_per_grid = (output_size + threads_per_block - 1) / threads_per_block;
  mat_add<<<blocks_per_grid, threads_per_block>>>(d_bias, dl_db, d_bias, 1,
                                                  output_size, neg, alpha);
  cuda_check(hipDeviceSynchronize());

  // return dl_dx;
  // FIXME: this is a temporary fix. cuda copying dl_dx into float *d_input
  // come up with a more graceful solution. This might become more aparent
  // once you start using tensors
  cuda_check(hipMemcpy(d_input, dl_dx, sizeof(float) * input_size,
                        hipMemcpyDeviceToDevice));
}

float *MlpLayer::get_host_weights() { return weights; }
float *MlpLayer::get_host_bias() { return bias; }
float *MlpLayer::get_device_weights() { return d_weights; }
float *MlpLayer::get_device_bias() { return d_bias; }
float *MlpLayer::get_weight_grad() { return dl_dw; }
float *MlpLayer::get_input_grad() { return dl_dx; }
float *MlpLayer::get_bias_grad() { return dl_db; }
float *MlpLayer::get_output_grad() { return dl_dz; }

// SoftmaxLayer

SoftmaxLayer::SoftmaxLayer(int input_size, int output_size)
    : input_size(input_size), output_size(output_size) {
  cuda_check(hipHostAlloc(&h_loss, sizeof(float), hipHostMallocDefault));
  cuda_check(
      hipHostAlloc(&y_hat, sizeof(float) * output_size, hipHostMallocDefault));
  cuda_check(
      hipHostAlloc(&y, sizeof(float) * output_size, hipHostMallocDefault));
  cuda_check(hipMalloc(&d_loss, sizeof(float)));
  cuda_check(hipMalloc(&dl_dx, input_size * sizeof(float)));
}

SoftmaxLayer::~SoftmaxLayer() {
  cuda_check(hipHostFree(h_loss));
  cuda_check(hipHostFree(y_hat));
  cuda_check(hipHostFree(y));
  cuda_check(hipFree(d_loss));
  cuda_check(hipFree(dl_dx));
}

int SoftmaxLayer::get_num_outputs() { return output_size; }
int SoftmaxLayer::get_num_inputs() { return input_size; }

void SoftmaxLayer::softmax(float *d_input, float *d_output, int batch_size) {
  int block_size = 128;
  int grid_size = (output_size + block_size - 1) / block_size;
  softmax_kernel<<<grid_size, block_size>>>(d_input, d_output, output_size);
}

void SoftmaxLayer::forward(float *d_y_hat, float *d_y, int batch_size) {
  cuda_check(hipMemcpy(y_hat, d_y_hat, sizeof(float) * output_size,
                        hipMemcpyDeviceToHost));
  cuda_check(
      hipMemcpy(y, d_y, sizeof(float) * output_size, hipMemcpyDeviceToHost));
  *h_loss = compute_cross_entropy_loss(y_hat, y, output_size);
  hipMemcpy(d_loss, h_loss, sizeof(float), hipMemcpyHostToDevice);
}

void SoftmaxLayer::back_prop(float *d_y_hat, float *d_y, float alpha) {
  // d_y_hat: device softmax output (probabilities) d_y: device one-hot label
  // dl_dx: device gradient buffer already allocated in ctor

  // Compute grad = d_y_hat - d_y into dl_dx
  int threads_per_block = 256;
  int blocks = (output_size + threads_per_block - 1) / threads_per_block;
  softmax_cross_entropy_backward<<<blocks, threads_per_block>>>(
      d_y_hat, d_y, dl_dx, output_size);
  cuda_check(hipDeviceSynchronize());
}
float SoftmaxLayer::get_loss() { return *h_loss; }

float *SoftmaxLayer::get_input_grad() { return dl_dx; }
