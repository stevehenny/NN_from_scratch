#include "hip/hip_runtime.h"
#include "cudaKernels.cuh"
#define BLOCK_SIZE 32
#define SHARED_BLOCK_SIZE BLOCK_SIZE + 2

__global__ void Convolution(float *A, float *B, float *C, int HA, int WA,
                            int HB, int WB, int HC, int WC, int input_channels,
                            int output_channels) {
  int col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
  int row = blockIdx.y * (BLOCK_SIZE - WC + 1) + threadIdx.y;
  int row_i = row - WC + 1;
  int col_i = col - WC + 1;

  float tmp = 0.0;

  // Declare shared memory for a tile of A
  __shared__ float shm[BLOCK_SIZE][BLOCK_SIZE];

  if (row_i < WA && row_i >= 0 && col_i < WA && col_i >= 0) {
    shm[threadIdx.y][threadIdx.x] = A[col_i * WA + row_i];
  } else {
    shm[threadIdx.y][threadIdx.x] = 0.0f;
  }

  __syncthreads();

  if (threadIdx.y < (BLOCK_SIZE - WC + 1) &&
      threadIdx.x < (BLOCK_SIZE - WC + 1) && row < (WB - WC + 1) &&
      col < (WB - WC + 1)) {
    for (int i = 0; i < WC; i++) {
      for (int j = 0; j < WC; j++) {
        tmp += shm[threadIdx.y + i][threadIdx.x + j] * C[j * WC + i];
      }
    }
    B[col * WB + row] = tmp;
  }
}

__global__ void Convolution3D(float *A, float *B, float *C, int HA, int WA,
                              int HB, int WB, int HC, int WC,
                              int input_channels, int output_channels) {
  int out_channel = blockIdx.z;

  // Global output location (row, col) this thread is responsible for
  int out_col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
  int out_row = blockIdx.y * (BLOCK_SIZE - HC + 1) + threadIdx.y;

  __shared__ float shm[SHARED_BLOCK_SIZE][SHARED_BLOCK_SIZE];

  float tmp = 0.0f;

  for (int in_channel = 0; in_channel < input_channels; ++in_channel) {
    float *input = A + in_channel * HA * WA;
    float *kernel = C + (out_channel * input_channels + in_channel) * HC * WC;

    // Global input coordinates this thread will load into shared memory
    int in_col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
    int in_row = blockIdx.y * (BLOCK_SIZE - HC + 1) + threadIdx.y;

    if (in_row < HA && in_col < WA && in_row >= 0 && in_col >= 0) {
      shm[threadIdx.y][threadIdx.x] = input[in_row * WA + in_col];
    } else {
      shm[threadIdx.y][threadIdx.x] = 0.0f;
    }

    __syncthreads();

    // Compute output only from threads assigned to compute one pixel
    if (threadIdx.y < (BLOCK_SIZE - HC + 1) &&
        threadIdx.x < (BLOCK_SIZE - WC + 1) && out_row < HB && out_col < WB) {

      for (int i = 0; i < HC; ++i) {
        for (int j = 0; j < WC; ++j) {
          tmp += shm[threadIdx.y + i][threadIdx.x + j] * kernel[i * WC + j];
        }
      }

      B[out_channel * HB * WB + out_row * WB + out_col] = tmp;
    }

    __syncthreads();
  }
}

__global__ void maxPool2D(float *A, float *B, int HA, int WA, int HB, int WB,
                          int input_channels) {
  int out_col = blockIdx.x * blockDim.x + threadIdx.x;
  int out_row = blockIdx.y * blockDim.y + threadIdx.y;
  int in_col = out_col * 2;
  int in_row = out_row * 2;
  int input_channel = blockIdx.z;
  __shared__ float shm[BLOCK_SIZE][BLOCK_SIZE];

  // populate shared memory
  for (int i = in_row; i <= in_row + 1; ++i) {
    for (int j = in_col; j <= in_col + 1; ++j) {
      shm[(i + in_row) - in_row][(j + in_col) - in_col] =
          A[input_channel * WA * HA + i * WA + j];
    }
  }
  __syncthreads();

  // Take the max of the data and write to B
  float temp = -1000.0f;
  float cur;
  for (int i = in_row; i <= in_row + 1; ++i) {
    for (int j = in_col; j <= in_col + 1; ++j) {
      cur = shm[(i + in_row) - in_row][(j + in_col) - in_col];
      if (temp < cur)
        temp = cur;
    }
  }
  __syncthreads();
  B[input_channel * WB * HB + out_row * WB + out_col] = temp;
}
