#include "CudaChecks.cuh"
#include "convLayer.cuh"
#include "cudaKernels.cuh"
#include <cstdint>
#include <iostream>
#include <stdexcept>
#define SHARED_BLOCK_SIZE BLOCK_SIZE + 2
convLayer::convLayer(ImageSize inputImageSize, ImageSize outputImageSize,
                     ImageSize kernelSize, float *kernels,
                     uint8_t input_channels, uint8_t output_channels)
    : input_channels(input_channels), output_channels(output_channels),
      kernels(kernels), HA(inputImageSize.height), WA(inputImageSize.width),
      HB(outputImageSize.height), WB(outputImageSize.width),
      HC(kernelSize.height), WC(kernelSize.width) {
  cudaCheck(hipMalloc((void **)&d_kernels, output_channels * KERNEL_SIZE *
                                                KERNEL_SIZE * sizeof(float)));
  cudaCheck(
      hipMemcpy(d_kernels, kernels,
                 output_channels * KERNEL_SIZE * KERNEL_SIZE * sizeof(float),
                 hipMemcpyHostToDevice));
}

convLayer::~convLayer() { hipFree(d_kernels); }

void convLayer::forward(float *input_image, float *output_image) {
  float *d_input_image;
  float *d_output_image;
  cudaCheck(hipMalloc((void **)&d_input_image,
                       input_channels * WA * HA * sizeof(float)));
  cudaCheck(hipMalloc((void **)&d_output_image,
                       output_channels * WB * HB * sizeof(float)));
  cudaCheck(hipMemcpy(d_input_image, input_image,
                       input_channels * WA * HA * sizeof(float),
                       hipMemcpyHostToDevice));

  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);

  int tile_output_size = BLOCK_SIZE - WC + 1;
  dim3 grid((WB + tile_output_size - 1) / tile_output_size,
            (HB + tile_output_size - 1) / tile_output_size, output_channels);

  Convolution3D<<<grid, threads>>>(d_input_image, d_output_image, d_kernels, HA,
                                   WA, HB, WB, HC, WC, input_channels,
                                   output_channels);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cerr << hipGetErrorString(error) << std::endl;
    throw std::runtime_error("Cuda kernel failed\n");
  }

  cudaCheck(hipDeviceSynchronize());
  cudaCheck(hipMemcpy(output_image, d_output_image,
                       output_channels * WB * HB * sizeof(float),
                       hipMemcpyDeviceToHost));

  cudaCheck(hipFree(d_input_image));
  cudaCheck(hipFree(d_output_image));
}
