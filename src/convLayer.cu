#include "hip/hip_runtime.h"
#include "CudaChecks.cuh"
#include "convLayer.cuh"
#include <cstdint>
#include <iostream>
#include <stdexcept>

// #define WA 28
// #define HA 28
// #define HC 3
// #define WC 3
// #define WB (WA - WC + 1)
// #define HB (HA - HC + 1)

__global__ void Convolution(float *A, float *B, float *C, int HA, int WA,
                            int HB, int WB, int HC, int WC) {
  int col = blockIdx.x * (BLOCK_SIZE - WC + 1) + threadIdx.x;
  int row = blockIdx.y * (BLOCK_SIZE - WC + 1) + threadIdx.y;
  int row_i = row - WC + 1;
  int col_i = col - WC + 1;

  float tmp = 0.0f;

  // Declare shared memory for a tile of A
  __shared__ float shm[BLOCK_SIZE][BLOCK_SIZE];

  if (row_i < WA && row_i >= 0 && col_i < WA && col_i >= 0) {
    shm[threadIdx.y][threadIdx.x] = A[col_i * WA + row_i];
  } else {
    shm[threadIdx.y][threadIdx.x] = 0.0f;
  }

  __syncthreads();

  if (threadIdx.y < (BLOCK_SIZE - WC + 1) &&
      threadIdx.x < (BLOCK_SIZE - WC + 1) && row < (WB - WC + 1) &&
      col < (WB - WC + 1)) {
    for (int i = 0; i < WC; i++) {
      for (int j = 0; j < WC; j++) {
        tmp += shm[threadIdx.y + i][threadIdx.x + j] * C[j * WC + i];
      }
    }
    B[col * WB + row] = tmp;
  }
}

convLayer::convLayer(ImageSize inputImageSize, ImageSize outputImageSize,
                     ImageSize kernelSize, float *kernels,
                     uint8_t input_channels, uint8_t output_channels)
    : input_channels(input_channels), output_channels(output_channels),
      kernels(kernels), HA(inputImageSize.height), WA(inputImageSize.width),
      HB(outputImageSize.height), WB(outputImageSize.width),
      HC(kernelSize.height), WC(kernelSize.width) {
  cudaCheck(hipMalloc((void **)&d_kernels, output_channels * KERNEL_SIZE *
                                                KERNEL_SIZE * sizeof(float)));
  cudaCheck(
      hipMemcpy(d_kernels, kernels,
                 output_channels * KERNEL_SIZE * KERNEL_SIZE * sizeof(float),
                 hipMemcpyHostToDevice));
}

convLayer::~convLayer() { hipFree(d_kernels); }

void convLayer::forward(float *input_image, float *output_image) {
  float *d_input_image;
  float *d_output_image;
  cudaCheck(hipMalloc((void **)&d_input_image, WA * HA * sizeof(float)));
  cudaCheck(hipMalloc((void **)&d_output_image, WB * HB * sizeof(float)));
  cudaCheck(hipMemcpy(d_input_image, input_image, WA * HA * sizeof(float),
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(d_output_image, output_image, WB * HB * sizeof(float),
                       hipMemcpyHostToDevice));

  dim3 threads(BLOCK_SIZE, BLOCK_SIZE);
  dim3 grid((WB - 1) / (BLOCK_SIZE - WC + 1), (WB - 1) / (BLOCK_SIZE - WC + 1),
            output_channels);

  Convolution<<<grid, threads>>>(d_input_image, d_output_image, d_kernels, HA,
                                 WA, HB, WB, HC, WC);

  hipError_t error = hipGetLastError();
  if (error != hipSuccess) {
    std::cerr << hipGetErrorString(error) << std::endl;
    throw std::runtime_error("Cuda kernel failed\n");
  }

  cudaCheck(hipDeviceSynchronize());
  cudaCheck(hipMemcpy(output_image, d_output_image, WB * HB * sizeof(float),
                       hipMemcpyDeviceToHost));

  cudaCheck(hipFree(d_input_image));
  cudaCheck(hipFree(d_output_image));
}
